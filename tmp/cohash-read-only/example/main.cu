/*
 *  (C) copyright  2011, Ismael Garcia, (U.Girona/ViRVIG, Spain & INRIA/ALICE, France)
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#ifndef EXAMPLE_01_CU_
#define EXAMPLE_01_CU_

#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <sys/syscall.h>
#include <sys/types.h>
#include <unistd.h>

#include <config_params.h>

#include <libh/hash.h>
#include <libhu/hash_utils.h>
#include <img_tga.inl>

#include <key_value_rand_hash.h>
#include <key_value_coh_hash.h>
#include <key_rand_hash.h>
#include <key_coh_hash.h>

#include <mt19937ar.h>

#if WIN32
#include <Windows.h>

struct timespec {
time_t tv_sec; /* Seconds since 00:00:00 GMT, */
               /* 1 January 1970 */
long tv_nsec; /* Additional nanoseconds since */
              /* tv_sec */
} timespec_t; 

LARGE_INTEGER
getFILETIMEoffset()
{
    SYSTEMTIME s;
    FILETIME f;
    LARGE_INTEGER t;
 
    s.wYear = 1970;
    s.wMonth = 1;
    s.wDay = 1;
    s.wHour = 0;
    s.wMinute = 0;
    s.wSecond = 0;
    s.wMilliseconds = 0;
    SystemTimeToFileTime(&s, &f);
    t.QuadPart = f.dwHighDateTime;
    t.QuadPart <<= 32;
    t.QuadPart |= f.dwLowDateTime;
    return (t);
}
 
int
clock_gettime(int X, struct timespec *tv)
{
    LARGE_INTEGER           t;
    FILETIME            f;
    double                  microseconds;
    static LARGE_INTEGER    offset;
    static double           frequencyToMicroseconds;
    static int              initialized = 0;
    static BOOL             usePerformanceCounter = 0;
 
    if (!initialized) {
        LARGE_INTEGER performanceFrequency;
        initialized = 1;
        usePerformanceCounter = QueryPerformanceFrequency(&performanceFrequency);
        if (usePerformanceCounter) {
            QueryPerformanceCounter(&offset);
            frequencyToMicroseconds = (double)performanceFrequency.QuadPart / 1000000.;
        } else {
            offset = getFILETIMEoffset();
            frequencyToMicroseconds = 10.;
        }
    }
    if (usePerformanceCounter) QueryPerformanceCounter(&t);
    else {
        GetSystemTimeAsFileTime(&f);
        t.QuadPart = f.dwHighDateTime;
        t.QuadPart <<= 32;
        t.QuadPart |= f.dwLowDateTime;
    }
 
    t.QuadPart -= offset.QuadPart;
    microseconds = (double)t.QuadPart / frequencyToMicroseconds;
    t.QuadPart = microseconds;
    tv->tv_sec = t.QuadPart / 1000000;
    tv->tv_nsec = (t.QuadPart % 1000000) * 1000;
    return (0);
}

#endif

//------------------------------------------------------------------------

#include <test_cu_robin_hood_hash.cu>

//------------------------------------------------------------------------

void runRobinHoodTest(ConfigParams& cfg) 
{
  testRobinHoodHash(cfg);
}

//------------------------------------------------------------------------

int runHashTest(int argc, char** argv, ConfigParams& cfg)
{
  if (argc < 6)
  {
    std::cerr << "* Num_threads                  -- " << OMP_CUSTOM_NUM_THREADS << std::endl;
    std::cerr << "* Random numbers hashing:"                                                                    << std::endl;
    std::cerr << "phash.exe [num_keys] [access_rate_non_valid_keys] [density] [seed] [access_mode]"             << std::endl;
    std::cerr << "  [num_keys]                   -- Integer number of keys"                                     << std::endl;
    std::cerr << "  [access_rate_non_valid_keys] -- Integer percentage 0-100"                                   << std::endl;
    std::cerr << "                                  (e.g. 20, means a total query of 20% non-valid"             << std::endl;
    std::cerr << "                                  accessed keys + [num_keys] of 80% valid keys)"              << std::endl;
    std::cerr << "  [density]                    -- Integer percentage 0-100"                                   << std::endl;
    std::cerr << "  [seed]                       -- Integer seed"                                               << std::endl;
    std::cerr << "  [access_mode]                -- 1 (sorted) / 0 (random shuffle)"                            << std::endl;
    std::cerr << std::endl;
    std::cerr << "(e.g. 'example_01.exe 32000000 100 80 61332125 1 -coh_hash')"                                 << std::endl;
    std::cerr << std::endl;
    std::cerr << "* Image data hashing:"                                                                        << std::endl;
    std::cerr << "phash.exe [image_file] [access_null_keys] [density] [seed] [access_mode]"                     << std::endl;
    std::cerr << "  [image_file]                 -- image data"                                                 << std::endl;
    std::cerr << "                                  (Use uncompressed tga images with RGBA"                     << std::endl;
    std::cerr << "                                   channels, value '#00000000' identify"                      << std::endl;
    std::cerr << "                                   non-valid key-data pixel entries)"                         << std::endl;
    std::cerr << "  [access_null_keys]           -- 1 access valid & non-valid keys /"                          << std::endl;
    std::cerr << "                                  0 access only valid keys"                                   << std::endl;
    std::cerr << "  [density]                    -- Integer percentage 0-100"                                   << std::endl;
    std::cerr << "  [seed]                       -- Integer seed"                                               << std::endl;
    std::cerr << "  [access_mode]                -- 1 (sorted) / 0 (random shuffle)"                            << std::endl;
    std::cerr << std::endl;
    std::cerr << "(e.g. 'example_01.exe flower_1024.tga 1 80 77016577 1 -coh_hash')"                            << std::endl;
    std::cerr << std::endl;
    
    return 0;
  }

  std::vector<std::string> sparams(argv, argv+argc);
  
  size_t found;
  found=sparams[1].find(".tga");
  if (found!=std::string::npos)
  {
    std::cerr << "image_data mode enabled" << std::endl;

    cfg.rand_num_mode     = false;
    cfg.image_mode        = true;
    cfg.image_name        = sparams[1];
    cfg.access_null_keys  = bool(atoi(sparams[2].c_str()));
    cfg.dens              = libhu::F32(atoi(sparams[3].c_str())) / 100.0f;
    cfg.seed              = atoi(sparams[4].c_str());
    cfg.sorted_access     = atoi(sparams[5].c_str());

    cfg.tex = loadTGA((char*)cfg.image_name.c_str());

    saveTGA(cfg.tex, "image_to_hash.tga");

    libhu::U32 tnnz = 0;
    for (libhu::U32 i = 0; i < cfg.tex->w * cfg.tex->h; i++)
    {
      libhu::U32 *imgPtr = (libhu::U32*)cfg.tex->data;
      if (imgPtr[i] != 0)
      {
        tnnz++;
      }
    }
    cfg.num_keys = tnnz;
    cfg.num_extra = cfg.tex->w * cfg.tex->h;
    if (cfg.access_null_keys)
    {
      cfg.rate_non_valid_keys = (float)(cfg.num_extra - cfg.num_keys) / (float)cfg.num_extra;
    }
    else
    {
      cfg.rate_non_valid_keys = 0.0;
      cfg.num_extra      = cfg.num_keys;
    }

  }
  else
  {
    cfg.rand_num_mode       = true;
    cfg.image_mode          = false;
    cfg.num_keys            = atoi(sparams[1].c_str());
    cfg.rate_non_valid_keys = (atoi(sparams[2].c_str()) / 100.0f);
    cfg.num_extra           = (cfg.num_keys / (1.0 - (atoi(sparams[2].c_str()) / 100.0f)));
    cfg.dens                = libhu::F32(atoi(sparams[3].c_str())) / 100.0f;
    cfg.seed                = atoi(sparams[4].c_str());
    cfg.sorted_access       = atoi(sparams[5].c_str());
  }
  
  if (argc > 6)
  {
    cfg.coh_hash          = (sparams[6] == "-coh_hash") ? 1 : 0;
    cfg.rand_hash         = (sparams[6] == "-rand_hash") ? 1 : 0;
  }
  else
  {
    cfg.coh_hash          = 1;
    cfg.rand_hash         = 1;
  }

  cfg.is_set = false;

  // Default 2D universe size
  cfg.u2D_w = 16384;
  cfg.u2D_h = 16384;

  // Default 3D universe size
  cfg.u3D_w = 512;
  cfg.u3D_h = 512;
  cfg.u3D_d = 512;

  runRobinHoodTest(cfg);

  libhu::F32 NUM_1M_KEYS = 1000000;
  libhu::F32 build_keys  = cfg.num_keys / NUM_1M_KEYS;
  libhu::F32 access_keys = ((cfg.rate_non_valid_keys == 0) ? cfg.num_keys : cfg.num_extra) / NUM_1M_KEYS;

  if (cfg.coh_hash)
  {
    std::cerr << "rh_coh_hash                      : " << cfg.rh_coh_hash_state << std::endl;
    std::cerr << "build rh_coh_hash                : " << std::setiosflags(std::ios::fixed) << std::setprecision(4) << cfg.rh_coh_hash_build_keys_per_sec << " Mkeys/sec" << std::endl;
    std::cerr << "access rh_coh_hash               : " << std::setiosflags(std::ios::fixed) << std::setprecision(4) << cfg.rh_coh_hash_access_keys_per_sec << " Mkeys/sec" << std::endl;
    std::cerr << std::endl;
    std::cerr << "-------------------------------------------" << std::endl;
  }
  else if (cfg.rand_hash)
  {
    std::cerr << "rh_rand_hash                     : " << cfg.rh_rand_hash_state << std::endl;
    std::cerr << "build rh_rand_hash               : " << std::setiosflags(std::ios::fixed) << std::setprecision(4) << cfg.rh_rand_hash_build_keys_per_sec << " Mkeys/sec" << std::endl;
    std::cerr << "access rh_rand_hash              : " << std::setiosflags(std::ios::fixed) << std::setprecision(4) << cfg.rh_rand_hash_access_keys_per_sec << " Mkeys/sec" << std::endl;
    std::cerr << std::endl;
    std::cerr << "-------------------------------------------" << std::endl;
  }

  if (cfg.image_mode)
  {
    delete cfg.tex;
  }

}

//------------------------------------------------------------------------

int main(int argc,char **argv)
{

  ConfigParams cfg;
  runHashTest(argc, argv, cfg);
  
  return 0;

}

//------------------------------------------------------------------------

#endif
